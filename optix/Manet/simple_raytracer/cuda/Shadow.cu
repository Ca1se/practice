#include "hip/hip_runtime.h"
#include "LaunchParams.h"
#include "Helper.h"

extern "C"
{
    __constant__ LaunchParams g_optix_launch_params;
}

extern "C" __global__ void __miss__shadow()
{
    float& payload = getPayload<float>();
    payload = 1.0f;
}