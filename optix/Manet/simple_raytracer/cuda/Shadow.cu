#include "hip/hip_runtime.h"
#include "LaunchParams.h"
#include "Helper.h"

extern "C" __constant__ LaunchParams g_optix_launch_params;

extern "C" __global__ void __miss__shadow()
{
    float4& payload = getPayload<float4>();
    payload = make_float4(1.0f, 0.0f, 0.0f, 1.0f);
}