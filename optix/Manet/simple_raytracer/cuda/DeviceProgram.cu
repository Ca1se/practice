#include "hip/hip_runtime.h"
#include <optix_device.h>
#include <VectorMath.h>

#include "LaunchParams.h"
#include "RecordData.h"
#include "Helper.h"

extern "C"
{
    __constant__ LaunchParams g_optix_launch_params;
}

extern "C" __global__ void
__closesthit__radiance()
{
    const HitgroupData* data = reinterpret_cast<HitgroupData*>(optixGetSbtDataPointer());

    const uint32_t primitive_id = optixGetPrimitiveIndex();
    const int3& nid = data->normal_indices[primitive_id];
    const int3& tid = data->texcoord_indices[primitive_id];

    const float2 barycentrics = optixGetTriangleBarycentrics();
    const float3 v0_normal = tputil::normalize(data->normals[nid.x]);
    const float3 v1_normal = tputil::normalize(data->normals[nid.y]);
    const float3 v2_normal = tputil::normalize(data->normals[nid.z]);
    const float3 face_normal = tputil::normalize(barycentrics.x * v0_normal + barycentrics.y * v1_normal + (1.0f - barycentrics.x - barycentrics.y) * v2_normal);

    const float3 ray_direction = optixGetWorldRayDirection();

    const float dotr = tputil::dot(ray_direction, face_normal);

    const float3 color = powf(fmaxf(0.0f, -dotr), 2.0f) * data->diffuse_color;

    float4& payload = getPayload<float4>();
    payload = make_float4(color, 1.0f);
}

extern "C" __global__ void
__anyhit__back_culling()
{
    const HitgroupData* data = reinterpret_cast<HitgroupData*>(optixGetSbtDataPointer());

    const uint32_t primitive_id = optixGetPrimitiveIndex();
    const int3& vid = data->vertex_indices[primitive_id];
    const float3& v0 = data->vertices[vid.x];
    const float3& v1 = data->vertices[vid.y];
    const float3& v2 = data->vertices[vid.z];
    const float3 ray_direction = optixGetWorldRayDirection();
    const float3 face_normal = tputil::cross(v2 - v0, v1 - v0);
    if(tputil::dot(ray_direction, face_normal) < 0.0f)
        optixIgnoreIntersection();
}

extern "C" __global__ void
__miss__radiance()
{
    const MissData* data = reinterpret_cast<MissData*>(optixGetSbtDataPointer());
    const float3& color   = data->background_color;

    float4& payload = getPayload<float4>();
    payload = make_float4(color, 1.0f);
}

static __device__ void
computeRay(float3& origin, float3& direction)
{
    const uint3 idx = optixGetLaunchIndex();
    const uint3 dim = optixGetLaunchDimensions();

    const float s = 2.0f * (static_cast<float>(idx.x) / static_cast<float>(dim.x)) - 1.0f;
    const float t = 2.0f * (static_cast<float>(idx.y) / static_cast<float>(dim.y)) - 1.0f;

    const auto& camera = g_optix_launch_params.camera;

    origin    = camera.position;
    direction = tputil::normalize(s * camera.u + t * camera.v + camera.w);
}

extern "C" __global__ void
__raygen__pinhole()
{
    const uint3 idx = optixGetLaunchIndex();
    const uint3 dim = optixGetLaunchDimensions();

    float3 origin;
    float3 direction;
    computeRay(origin, direction);

    float4 color;
    uint32_t i0, i1;
    packPointer(static_cast<void*>(&color), i0, i1);

    optixTrace(g_optix_launch_params.handle,
               origin,
               direction,
               0.0f,
               1e16f,
               0.0f,
               OptixVisibilityMask{ 255 },
               OPTIX_RAY_FLAG_NONE,
               0,
               1,
               0,
               i0,
               i1);

    const size_t pixel_index = dim.x * idx.y + idx.x;

    g_optix_launch_params.frame.color_buffer[pixel_index] = make_uchar4(255.99f * color);
}