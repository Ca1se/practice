#include "hip/hip_runtime.h"
#include <optix_device.h>
#include <VectorMath.h>

#include "../RecordData.h"
#include "LaunchParams.h"
#include "Helper.h"

extern "C"
{
    __constant__ LaunchParams g_optix_launch_params;
}

extern "C" __global__ void
__closesthit__radiance()
{
    const HitgroupData* data = reinterpret_cast<HitgroupData*>(optixGetSbtDataPointer());

    const uint32_t primitive_id = optixGetPrimitiveIndex();
    const uint3& vid = data->indices[primitive_id];
    const float3& A  = data->vertices[vid.x];
    const float3& B  = data->vertices[vid.y];
    const float3& C  = data->vertices[vid.z];

    const float3 ray_direction = optixGetWorldRayDirection();
    const float3 face_normal   = tputil::normalize(tputil::cross(C - A, B - A));

    const float3 gray_color = make_float3(211.0f, 211.0f, 211.0f);
    const float dotr = tputil::dot(ray_direction, face_normal);

    const float3 color = fmaxf(0.0f, -dotr) * gray_color;

    uchar4& payload = getPayload<uchar4>();
    payload = make_uchar4(static_cast<uint8_t>(color.x),
                          static_cast<uint8_t>(color.y),
                          static_cast<uint8_t>(color.z),
                          0xff);
}

extern "C" __global__ void
__miss__radiance()
{
    MissData* data = reinterpret_cast<MissData*>(optixGetSbtDataPointer());
    uchar3 color   = data->background_color;

    uchar4& payload = getPayload<uchar4>();
    payload = make_uchar4(color.x, color.y, color.z, 0xff);
}

static __device__ void
computeRay(float3& origin, float3& direction)
{
    const uint3 idx = optixGetLaunchIndex();
    const uint3 dim = optixGetLaunchDimensions();

    const float s = 2.0f * (static_cast<float>(idx.x) / static_cast<float>(dim.x)) - 1.0f;
    const float t = 2.0f * (static_cast<float>(idx.y) / static_cast<float>(dim.y)) - 1.0f;

    const auto& camera = g_optix_launch_params.camera;

    origin    = camera.position;
    direction = tputil::normalize(s * camera.u + t * camera.v + camera.w);
}

extern "C" __global__ void
__raygen__pinhole()
{
    const uint3 idx = optixGetLaunchIndex();
    const uint3 dim = optixGetLaunchDimensions();

    float3 origin;
    float3 direction;
    computeRay(origin, direction);

    uchar4 color;
    uint32_t i0, i1;
    packPointer(static_cast<void*>(&color), i0, i1);

    optixTrace(g_optix_launch_params.handle,
               origin,
               direction,
               0.0f,
               1e16f,
               0.0f,
               OptixVisibilityMask{ 255 },
               OPTIX_RAY_FLAG_NONE,
               0,
               1,
               0,
               i0,
               i1);

    const size_t pixel_index = dim.x * idx.y + idx.x;

    g_optix_launch_params.frame.color_buffer[pixel_index] = color;
}