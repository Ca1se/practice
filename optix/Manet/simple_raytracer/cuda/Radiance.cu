#include "hip/hip_runtime.h"
#include <optix_device.h>
#include <VectorMath.h>

#include "LaunchParams.h"
#include "Helper.h"

extern "C"
{
    __constant__ LaunchParams g_optix_launch_params;
}

extern "C" __global__ void __closesthit__radiance()
{
    const HitgroupData* data = reinterpret_cast<HitgroupData*>(optixGetSbtDataPointer());

    const uint32_t primitive_id = optixGetPrimitiveIndex();
    const int3& vid = data->vertex_indices[primitive_id];
    const int3& nid = data->normal_indices[primitive_id];
    const int3& tid = data->texcoord_indices[primitive_id];

    const float2 barycentrics = optixGetTriangleBarycentrics();
    const float b1 = barycentrics.x;
    const float b2 = barycentrics.y;
    const float b0 = 1.0f - b1 - b2;

    const float3 ray_direction = optixGetWorldRayDirection();
    const float3 intersection = optixGetWorldRayOrigin() + optixGetRayTmax() * ray_direction;
    const float3 normal   = nid.x != -1
                            ? tputil::normalize(b0 * data->normals[nid.x]
                                              + b1 * data->normals[nid.y]
                                              + b2 * data->normals[nid.z])
                            : plainNormal(data->vertices[vid.x], data->vertices[vid.y], data->vertices[vid.z]);
    const float2 texcoord = tid.x != -1
                            ? b0 * data->texcoords[tid.x]
                            + b1 * data->texcoords[tid.y]
                            + b2 * data->texcoords[tid.z]
                            : make_float2(0.0f, 0.0f);

    const float3 light_position = make_float3(115.57f, 923.70f, -11.59f);
    const float3 shadow_direction = light_position - intersection;

    float light_visibility = 0.0f;
    uint32_t i0, i1;
    packPointer(static_cast<void*>(&light_visibility), i0, i1);

    optixTrace(g_optix_launch_params.handle,
               intersection,
               shadow_direction,
               0.001f,
               1.0f - 0.001f,
               0.0f,
               OptixVisibilityMask{ 255 },
               OPTIX_RAY_FLAG_TERMINATE_ON_FIRST_HIT 
               | OPTIX_RAY_FLAG_DISABLE_ANYHIT 
               | OPTIX_RAY_FLAG_DISABLE_CLOSESTHIT,
               0,
               1,
               1,
               i0,
               i1);
    
    float3 color = data->diffuse_color;
    if(data->texture != 0) {
        const float4 tex = tex2D<float4>(data->texture, texcoord.x, texcoord.y);
        color *= make_float3(tex.x, tex.y, tex.z);
    }

    const float cosdn = 0.1f + 0.8f * -tputil::dot(ray_direction, normal);
    color = (0.2f + (0.2f + 0.8f * light_visibility) * cosdn) * color;

    float4& payload = getPayload<float4>();
    payload = make_float4(color, 1.0f);
}

extern "C" __global__ void __anyhit__back_culling()
{
    const HitgroupData* data = reinterpret_cast<HitgroupData*>(optixGetSbtDataPointer());

    const uint32_t primitive_id = optixGetPrimitiveIndex();
    const int3& vid = data->vertex_indices[primitive_id];
    const float3 normal = plainNormal(data->vertices[vid.x], data->vertices[vid.y], data->vertices[vid.z]);
    const float3 ray_direction = optixGetWorldRayDirection();
    if (tputil::dot(normal, ray_direction) >= 0.0f)
        optixIgnoreIntersection();
}

extern "C" __global__ void __miss__radiance()
{
    float4& payload = getPayload<float4>();
    payload = make_float4(0.5f, 0.7f, 1.0f, 1.0f);
}

extern "C" __global__ void __raygen__pinhole()
{
    const uint3 idx    = optixGetLaunchIndex();
    const uint3 dim    = optixGetLaunchDimensions();
    const auto& camera = g_optix_launch_params.camera;
    const uint32_t accum_id = g_optix_launch_params.frame.accum_id;
    const size_t pixel_index = dim.x * idx.y + idx.x;

    // Get ray direction
    uint32_t seed = tea<4>(pixel_index, accum_id);
    const float2 subpixel_jitter = accum_id == 0 ? make_float2(0.5f, 0.5f) : make_float2(rnd(seed), rnd(seed));
    const float2 st = 2.0f
                    * make_float2((static_cast<float>(idx.x) + subpixel_jitter.x) / static_cast<float>(dim.x),
                                  (static_cast<float>(idx.y) + subpixel_jitter.y) / static_cast<float>(dim.y))
                    - 1.0f;
    const float3 ray_origin    = g_optix_launch_params.camera.position;
    const float3 ray_direction = tputil::normalize(st.x * camera.u + st.y * camera.v + camera.w);

    // Pack payload pointer to 2 uints;
    float4 color;
    uint32_t i0, i1;
    packPointer(static_cast<void*>(&color), i0, i1);

    optixTrace(g_optix_launch_params.handle,
               ray_origin,
               ray_direction,
               0.0f,
               1e16f,
               0.0f,
               OptixVisibilityMask{ 255 },
               OPTIX_RAY_FLAG_CULL_BACK_FACING_TRIANGLES,
               0,
               1,
               0,
               i0,
               i1);

    uchar4& output_color = g_optix_launch_params.frame.color_buffer[pixel_index];
    float4& accum_color  = g_optix_launch_params.frame.accum_buffer[pixel_index];

    accum_color += color;
    if(accum_id == 0)
        accum_color = color;
    output_color = make_uchar4(255.99f * (accum_color / static_cast<float>(accum_id + 1)));
}