#include "hip/hip_runtime.h"
#include <optix_device.h>
#include <VectorMath.h>
#include <>

#include "LaunchParams.h"
#include "Helper.h"

extern "C" __constant__ LaunchParams g_optix_launch_params;

extern "C" __global__ void __closesthit__radiance()
{
    const HitgroupData* data = reinterpret_cast<HitgroupData*>(optixGetSbtDataPointer());

    const uint32_t primitive_id = optixGetPrimitiveIndex();
    const int3& vid = data->vertex_indices[primitive_id];
    const int3& nid = data->normal_indices[primitive_id];
    const int3& tid = data->texcoord_indices[primitive_id];

    const float2 barycentrics = optixGetTriangleBarycentrics();
    const float b1 = barycentrics.x;
    const float b2 = barycentrics.y;
    const float b0 = 1.0f - b1 - b2;

    const float3 ray_direction = optixGetWorldRayDirection();
    const float3 normal   = tputil::normalize(b0 * data->normals[nid.x]
                                            + b1 * data->normals[nid.y]
                                            + b2 * data->normals[nid.z]);
    const float2 texcoord = b0 * data->texcoords[tid.x]
                          + b1 * data->texcoords[tid.y]
                          + b2 * data->texcoords[tid.z];
    const float dotv = tputil::dot(ray_direction, normal);

    float3 color = data->diffuse_color;
    if(data->texture != 0) {
        const float4 tex = tex2D<float4>(data->texture, texcoord.x, texcoord.y);
        color *= make_float3(tex.x, tex.y, tex.z);
    }

    const float factor = 0.2f + 0.8f * fmaxf(0.0f, -dotv);

    float4& payload = getPayload<float4>();
    payload = make_float4(factor * color, 1.0f);
}

extern "C" __global__ void __miss__radiance()
{
    float4& payload = getPayload<float4>();
    payload = make_float4(0.5f, 0.7f, 1.0f, 1.0f);
}

extern "C" __global__ void __raygen__pinhole()
{
    const uint3 idx = optixGetLaunchIndex();
    const uint3 dim = optixGetLaunchDimensions();

    float3 origin;
    float3 direction;
    computeRay(g_optix_launch_params.camera, origin, direction);

    float4 color;
    uint32_t i0, i1;
    packPointer(static_cast<void*>(&color), i0, i1);

    optixTrace(g_optix_launch_params.handle,
               origin,
               direction,
               0.0f,
               1e16f,
               0.0f,
               OptixVisibilityMask{ 255 },
               OPTIX_RAY_FLAG_CULL_BACK_FACING_TRIANGLES,
               0,
               1,
               0,
               i0,
               i1);

    const size_t pixel_index = dim.x * idx.y + idx.x;

    g_optix_launch_params.frame.color_buffer[pixel_index] = make_uchar4(255.99f * color);
}