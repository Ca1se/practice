#include "hip/hip_runtime.h"
#include <VectorMath.h>
#include <optix_device.h>

#include "../RecordData.h"
#include "LaunchParams.h"

extern "C"
{
    __constant__ LaunchParams g_optix_launch_params;
}

extern "C" __global__ void
__closesthit__radiance()
{
    HitgroupData* data = reinterpret_cast<HitgroupData*>(optixGetSbtDataPointer());
    uchar3 color = data->color;

}

extern "C" __global__ void
__miss__radiance()
{
    MissData* data = reinterpret_cast<MissData*>(optixGetSbtDataPointer());
    uchar3 color   = data->background_color;

    optixSetPayload_0(color.x);
    optixSetPayload_1(color.y);
    optixSetPayload_2(color.z);
}

static __device__ void
computeRay(float3& origin, float3& direction)
{
    const uint3 idx = optixGetLaunchIndex();
    const uint3 dim = optixGetLaunchDimensions();

    const float s = 2.0f * (static_cast<float>(idx.x) / static_cast<float>(dim.x)) - 1.0f;
    const float t = 2.0f * (static_cast<float>(idx.y) / static_cast<float>(dim.y)) - 1.0f;

    const auto& camera = g_optix_launch_params.camera;

    origin    = camera.position;
    direction = tputil::normalize(s * camera.u + t * camera.v + camera.w);
}

extern "C" __global__ void
__raygen__pinhole()
{
    const uint3 idx = optixGetLaunchIndex();
    const uint3 dim = optixGetLaunchDimensions();

    float3 origin;
    float3 direction;
    computeRay(origin, direction);

    uint32_t r, g, b;
    optixTrace(g_optix_launch_params.handle,
               origin,
               direction,
               0.0f,
               1e16f,
               0.0f,
               OptixVisibilityMask{ 255 },
               OPTIX_RAY_FLAG_NONE,
               0,
               1,
               0,
               r,
               g,
               b);

    const size_t pixel_index = dim.x * idx.y + idx.x;

    g_optix_launch_params.frame.color_buffer[pixel_index] = make_uchar4(r, g, b, 0xff);
}