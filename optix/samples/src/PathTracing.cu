#include "hip/hip_runtime.h"
#include "PathTracing.h"

#include <hip/hip_vector_types.h>
#include <optix_device.h>

#include "Manet.h"
#include "random.h"
#include "Helper.h"
#include "Record.h"
#include "HitResult.h"

namespace
{

__constant__ struct
{
    __forceinline__ __device__ OptixPayloadTypeID constexpr payloadType(PayloadType payload_type)
    {
        switch (payload_type) {
        case PAYLOAD_TYPE_RADIANCE:
            return OPTIX_PAYLOAD_TYPE_ID_0;
        case PAYLOAD_TYPE_OCCLUSION:
            return OPTIX_PAYLOAD_TYPE_ID_1;
        }
    }

} g_mapper;

__forceinline__ __device__ void setPayloadType(PayloadType payload_type)
{
    optixSetPayloadTypes(g_mapper.payloadType(payload_type));
}

__forceinline__ __device__ RadiancePayload& getRadiancePayload()
{
    setPayloadType(PAYLOAD_TYPE_RADIANCE);
    return getPayload<RadiancePayload&>();
}

__forceinline__ __device__ void
traceRadiance(OptixTraversableHandle handle,
              float                  tmin,
              float                  tmax,
              RadiancePayload&       payload)
{
    uint32_t u0, u1;
    packPointer(u0, u1, &payload);

    optixTrace(g_mapper.payloadType(PAYLOAD_TYPE_RADIANCE),
               handle,
               payload.ray_origin,
               payload.ray_direction,
               tmin,
               tmax,
               0.0f,  // ray time
               1,     // visibility mask
               OPTIX_RAY_FLAG_DISABLE_ANYHIT | OPTIX_RAY_FLAG_CULL_BACK_FACING_TRIANGLES,
               RAY_TYPE_RADIANCE,
               RAY_TYPE_COUNT,
               RAY_TYPE_RADIANCE,
               u0, u1);
}

} // namespace

extern "C" __constant__ LaunchParams g_launch_params;

extern "C" __global__ void __raygen__pinhole()
{
    const uint3    idx               = optixGetLaunchIndex();
    const uint3    dim               = optixGetLaunchDimensions();
    const size_t   pixel_index       = dim.x * idx.y + idx.x;
    const uint32_t accum_count       = g_launch_params.frame.accum_count;
    const uint32_t samples_per_pixel = g_launch_params.samples_per_pixel;
    const uint32_t max_tracing_num   = g_launch_params.max_tracing_num;
    const auto&    camera            = g_launch_params.camera;

    uint32_t seed = tea<4>(pixel_index, accum_count);
    float3 result = make_float3(0.0f);

    for (uint32_t i = 0; i < samples_per_pixel; i++) {
        const float2 subpixel_jitter = make_float2(rnd(seed), rnd(seed));
        const float2 st = 2.0f
                        * make_float2((static_cast<float>(idx.x) + subpixel_jitter.x) / static_cast<float>(dim.x),
                                    (static_cast<float>(idx.y) + subpixel_jitter.y) / static_cast<float>(dim.y))
                        - 1.0f;
        RadiancePayload payload = {};
        payload.ray_origin    = camera.position;
        payload.ray_direction = normalize(st.x * camera.u + st.y * camera.v - camera.w);
        payload.seed          = seed;

        do {
            traceRadiance(g_launch_params.handle, 0.0f, 1e16f, payload);
            result += payload.attenuation * payload.radiance;
            payload.depth++;
        } while (!payload.done || payload.depth >= max_tracing_num);
    }

    float4& accum_color  = g_launch_params.frame.accum_buffer[pixel_index];
    uchar4& pixel_color  = g_launch_params.frame.color_buffer[pixel_index];
    float3  result_color = result / static_cast<float>(samples_per_pixel);

    if (accum_count > 0) {
        const float factor = 1.0f / static_cast<float>(accum_count + 1);
        result_color = lerp(make_float3(accum_color), result_color, factor);
    }

    accum_color = make_float4(result_color, 1.0f);
    pixel_color = make_color(result_color);
}

extern "C" __global__ void __closesthit__radiance()
{
    const HitgroupData* data     = reinterpret_cast<HitgroupData*>(optixGetSbtDataPointer());
    const HitResult     result   = getHitResult(*data);
    const PbrMaterial&  material = *(data->material);
    RadiancePayload&    payload  = getRadiancePayload();

    float4 base_color = material.base_color * result.color;
    if (material.base_color_texture) {
        const float4 texture_color        = sampleTexture<float4>(material.base_color_texture, result);
        const float3 texture_color_linear = linearize(make_float3(texture_color));
        base_color *= make_float4(texture_color_linear, texture_color.w);
    }

    float metallic  = material.metallic;
    float roughness = material.roughness;
    if (material.metallic_roughness_texture) {
        const float4 metallic_roughness = sampleTexture<float4>(material.metallic_roughness_texture, result);
        metallic  *= metallic_roughness.z;
        roughness *= metallic_roughness.y;
    }

    const float3 albedo         = make_float3(base_color);
    const float3 f0             = make_float3(0.04f);
    const float3 diffuse_color  = albedo * (1.0f - metallic);
    const float3 specular_color = lerp(f0, albedo, metallic);

    float3 radiance = make_float3(0.0f);

    // emission
    if (material.emissive_texture) {
        const float3 emissive_factor = material.emissive_factor;
        const float4 emissive_texture_color = sampleTexture<float4>(material.emissive_texture, result);
        radiance += emissive_factor * make_float3(emissive_texture_color);
    }

    // direct lighting
    float3 normal = result.normal;
    if (material.normal_texture) {
        const float4 normal_sampled = 2.0f * sampleTexture<float4>(material.normal_texture, result) - make_float4(1.0f);
        const float2 rotation       = material.normal_texture.texcoord_rotation;
        const float2 tb             = make_float2(normal_sampled.x, normal_sampled.y);
        const float2 tb_trans       = make_float2(dot(tb, make_float2(rotation.y, -rotation.x)),
                                                  dot(tb, make_float2(rotation.x, rotation.y)));
        normal = normalize(tb_trans.x * result.texcoord.t + tb_trans.y * result.texcoord.b + normal_sampled.z * result.normal);
    }
}

extern "C" __global__ void __miss__radiance()
{
    RadiancePayload& payload = getRadiancePayload();

    payload.radiance = make_float3(0.0f); // if depth > 0 and ray missed, then no radiance
    if (payload.depth == 0) // directly hit the background
        payload.radiance = g_launch_params.background_color;

    payload.done = true;
}

extern "C" __global__ void __closesthit__occlusion()
{
}

extern "C" __global__ void __miss__occlusion()
{
}