#include "hip/hip_runtime.h"
#include "PathTracing.h"

#include <hip/hip_vector_types.h>
#include <optix_device.h>

#include "Manet.h"
#include "random.h"
#include "Helper.h"
#include "Record.h"
#include "HitResult.h"

namespace
{

struct Onb
{
    __forceinline__ __device__ Onb(const float3& normal)
    {
        m_normal = normal;

        if(fabs(m_normal.x) > fabs(m_normal.z))
        {
            m_binormal.x = -m_normal.y;
            m_binormal.y =  m_normal.x;
            m_binormal.z =  0;
        }
        else
        {
            m_binormal.x =  0;
            m_binormal.y = -m_normal.z;
            m_binormal.z =  m_normal.y;
        }

        m_binormal = normalize(m_binormal);
        m_tangent = cross(m_binormal, m_normal);
    }

    __forceinline__ __device__ void inverse_transform(float3& p) const
    {
        p = p.x * m_tangent + p.y * m_binormal + p.z * m_normal;
    }

    float3 m_tangent;
    float3 m_binormal;
    float3 m_normal;
};

__constant__ struct
{
    __forceinline__ __device__ OptixPayloadTypeID constexpr payloadType(PayloadType payload_type)
    {
        switch (payload_type) {
        case PAYLOAD_TYPE_RADIANCE:
            return OPTIX_PAYLOAD_TYPE_ID_0;
        case PAYLOAD_TYPE_OCCLUSION:
            return OPTIX_PAYLOAD_TYPE_ID_1;
        }
    }

} g_mapper;

__forceinline__ __device__ RadiancePayload& getRadiancePayload()
{
    return getPayload<RadiancePayload&>();
}

__forceinline__ __device__ void
traceRadiance(OptixTraversableHandle handle,
              float                  tmin,
              float                  tmax,
              RadiancePayload&       payload)
{
    uint32_t u0, u1;
    packPointer(u0, u1, &payload);

    optixTrace(g_mapper.payloadType(PAYLOAD_TYPE_RADIANCE),
               handle,
               payload.ray_origin,
               payload.ray_direction,
               tmin,
               tmax,
               0.0f,  // ray time
               OptixVisibilityMask{ 255 },    // visibility mask
               OPTIX_RAY_FLAG_DISABLE_ANYHIT | OPTIX_RAY_FLAG_CULL_BACK_FACING_TRIANGLES,
               // OPTIX_RAY_FLAG_NONE,
               RAY_TYPE_RADIANCE,
               RAY_TYPE_COUNT,
               RAY_TYPE_RADIANCE,
               u0, u1);
}

__forceinline__ __device__ bool
traceOcclusion(OptixTraversableHandle handle,
               float3                 ray_origin,
               float3                 ray_direction,
               float                  tmin,
               float                  tmax)
{
    uint32_t occluded;
    optixTrace(g_mapper.payloadType(PAYLOAD_TYPE_OCCLUSION),
               handle,
               ray_origin,
               ray_direction,
               tmin,
               tmax,
               0.0f, // ray time
               OptixVisibilityMask{ 255 },    // visibility mask
               OPTIX_RAY_FLAG_DISABLE_ANYHIT | OPTIX_RAY_FLAG_TERMINATE_ON_FIRST_HIT,
               // OPTIX_RAY_FLAG_TERMINATE_ON_FIRST_HIT,
               RAY_TYPE_OCCLUSION,
               RAY_TYPE_COUNT,
               RAY_TYPE_OCCLUSION,
               occluded);
    return static_cast<bool>(occluded);
}

} // namespace

extern "C" __constant__ LaunchParams g_launch_params;

extern "C" __global__ void __raygen__pinhole()
{
    const uint3    idx               = optixGetLaunchIndex();
    const uint3    dim               = optixGetLaunchDimensions();
    const size_t   pixel_index       = dim.x * idx.y + idx.x;
    const uint32_t accum_count       = g_launch_params.frame.accum_count;
    const uint32_t samples_per_pixel = g_launch_params.samples_per_pixel;
    const uint32_t max_tracing_num   = g_launch_params.max_tracing_num;
    const auto     camera            = g_launch_params.camera;

    uint32_t seed = tea<4>(pixel_index, accum_count);
    float3 result = make_float3(0.0f);

    for (uint32_t i = 0; i < samples_per_pixel; i++) {
        const float2 subpixel_jitter = make_float2(rnd(seed), rnd(seed));
        const float2 st = 2.0f
                          * make_float2((static_cast<float>(idx.x) + subpixel_jitter.x) / static_cast<float>(dim.x),
                                        (static_cast<float>(idx.y) + subpixel_jitter.y) / static_cast<float>(dim.y))
                          - 1.0f;
        RadiancePayload payload = {};
        payload.ray_origin    = camera.position;
        payload.ray_direction = normalize(st.x * camera.u + st.y * camera.v - camera.w);
        payload.seed          = seed;

        do {
            traceRadiance(g_launch_params.handle, 0.01f, 1e16f, payload);
            result += payload.attenuation * payload.radiance;
            payload.depth++;
        } while (!payload.done && payload.depth < max_tracing_num);
    }

    float3 result_color = result / static_cast<float>(samples_per_pixel);
    float4 accum_color  = g_launch_params.frame.accum_buffer[pixel_index];

    if (accum_count > 0) {
        const float factor = 1.0f / static_cast<float>(accum_count + 1);
        result_color = clamp(lerp(make_float3(accum_color), result_color, factor), make_float3(0.0f), make_float3(1.0f));
    }

    g_launch_params.frame.accum_buffer[pixel_index] = make_float4(result_color, 1.0f);
    g_launch_params.frame.color_buffer[pixel_index] = make_color(result_color);
}

extern "C" __global__ void __closesthit__radiance()
{
    optixSetPayloadTypes(PAYLOAD_TYPE_RADIANCE);

    const HitgroupData*       data     = reinterpret_cast<HitgroupData*>(optixGetSbtDataPointer());
    const HitResult           result   = getHitResult(*data);
    const PbrMaterial&        material = *(data->material);
    const ParallelogramLight& light    = g_launch_params.light;
    RadiancePayload&          payload  = getRadiancePayload();
    uint32_t                  seed     = payload.seed;

    float4 base_color = material.base_color * result.color;
    if (material.base_color_texture) {
        const float4 texture_color        = sampleTexture<float4>(material.base_color_texture, result);
        const float3 texture_color_linear = linearize(make_float3(texture_color));
        base_color *= make_float4(texture_color_linear, texture_color.w);
    }

    float3 radiance    = make_float3(0.0f);
    float3 attenuation = make_float3(1.0f);

    // emission
    if (material.emissive_texture) {
        const float3 emissive_factor = material.emissive_factor;
        const float4 emissive_texture_color = sampleTexture<float4>(material.emissive_texture, result);
        radiance += emissive_factor * make_float3(emissive_texture_color);
    }

    // normal
    float3 normal = result.normal;
    if (material.normal_texture) {
        const float4 normal_sampled = 2.0f * sampleTexture<float4>(material.normal_texture, result) - make_float4(1.0f);
        const float2 rotation       = material.normal_texture.texcoord_rotation;
        const float2 tb             = make_float2(normal_sampled.x, normal_sampled.y);
        const float2 tb_trans       = make_float2(dot(tb, make_float2(rotation.y, -rotation.x)),
                                                  dot(tb, make_float2(rotation.x, rotation.y)));
        normal = normalize(tb_trans.x * result.texcoord.t + tb_trans.y * result.texcoord.b + normal_sampled.z * result.normal);
    }

    if (dot(normal, payload.ray_direction) > 0.0f)
        normal = -normal;

    float metallic  = material.metallic;
    float roughness = material.roughness;
    if (material.metallic_roughness_texture) {
        const float4 metallic_roughness = sampleTexture<float4>(material.metallic_roughness_texture, result);
        metallic  *= metallic_roughness.z;
        roughness *= metallic_roughness.y;
    }
    
    // BRDF
    const float  s = 2.0f * rnd(seed) - 1.0f;
    const float  t = 2.0f * rnd(seed) - 1.0f;
    const float3 sample_pos = light.center + s * light.half_u + t * light.half_v;
    const float3 sample_dir = normalize(sample_pos - result.intersection);
    const float3 half_vec   = normalize(sample_dir - payload.ray_direction);
    const float  N_dot_L    = dot(normal, sample_dir);
    const float  N_dot_V    = dot(normal, -payload.ray_direction);
    const float  N_dot_H    = dot(normal, half_vec);
    const float  V_dot_H    = dot(-payload.ray_direction, half_vec);

    const float  alpha  = roughness * roughness;
    const float3 albedo = make_float3(base_color);
    const float3 f0     = lerp(make_float3(0.04f), albedo, metallic);
    const float3 F      = schlick(f0, V_dot_H);
    const float  G      = smiths(N_dot_V, N_dot_L, roughness);
    const float  D      = ggxNormal(N_dot_H, alpha);
    const float3 ks     = F;
    const float3 kd     = (make_float3(1.0f) - ks) * (1.0f - metallic);
    
    const float3 f_r = kd * albedo / MANET_PI + ks * 0.25f * G * D / (N_dot_V * N_dot_L) * F;

    attenuation *= f_r * N_dot_L * 2.0f * MANET_PI / g_launch_params.p_rr;

    /*
    // direct light
    if (dot(normal, sample_dir) > 0.0f && dot(normal, -payload.ray_direction) > 0.0f) {
        const float distance = length(sample_pos - result.intersection);
        const bool  occluded = traceOcclusion(g_launch_params.handle,
                                              result.intersection,
                                              sample_dir,
                                              0.01f,
                                              distance - 0.01f);
                                            
        if (!occluded) {
            const float area = 4.0f * length(light.half_u) * length(light.half_v);
            radiance += light.emission * f_r * N_dot_L * dot(light.normal, -sample_dir) * area / (distance * distance);
        }
    }
    */

    payload.radiance = radiance;
    payload.attenuation *= attenuation;

    if (rnd(seed) > g_launch_params.p_rr) {
        payload.done = true;
        return;
    }

    // uniform hemisphere sample
    float3 sample_vec;
    cosine_sample_hemisphere(rnd(seed), rnd(seed), sample_vec);
    Onb onb(normal);
    onb.inverse_transform(sample_vec);

    payload.ray_origin    = result.intersection;
    payload.ray_direction = normalize(sample_vec);
}

extern "C" __global__ void __miss__radiance()
{
    optixSetPayloadTypes(PAYLOAD_TYPE_RADIANCE);
    RadiancePayload& payload = getRadiancePayload();

    payload.radiance = make_float3(0.0f); // if depth > 0 and ray missed, then no radiance
    if (payload.depth == 0) // directly hit the background
        payload.radiance = g_launch_params.background_color;

    payload.done = true;
}

extern "C" __global__ void __closesthit__occlusion()
{
    optixSetPayloadTypes(PAYLOAD_TYPE_OCCLUSION);
    optixSetPayload_0(1);
}

extern "C" __global__ void __miss__occlusion()
{
    optixSetPayloadTypes(PAYLOAD_TYPE_OCCLUSION);
    optixSetPayload_0(0);
}