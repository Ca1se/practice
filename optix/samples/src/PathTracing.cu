#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>
#include <optix_device.h>

#include "Manet.h"
#include "random.h"
#include "Helper.h"
#include "Record.h"
#include "HitResult.h"

extern "C" __constant__ LaunchParams g_launch_params;


extern "C" __global__ void __raygen__pinhole()
{
    const uint3    idx         = optixGetLaunchIndex();
    const uint3    dim         = optixGetLaunchDimensions();
    const size_t   pixel_index = dim.x * idx.y + idx.x;
    const uint32_t accum_count = g_launch_params.frame.accum_count;
    const auto&    camera      = g_launch_params.camera;

    // Get ray direction
    uint32_t seed = tea<4>(pixel_index, accum_count);
    const float2 subpixel_jitter = accum_count == 0 ? make_float2(0.5f, 0.5f) : make_float2(rnd(seed), rnd(seed));
    const float2 st = 2.0f
                    * make_float2((static_cast<float>(idx.x) + subpixel_jitter.x) / static_cast<float>(dim.x),
                                  (static_cast<float>(idx.y) + subpixel_jitter.y) / static_cast<float>(dim.y))
                    - 1.0f;
    const float3 ray_origin    = camera.position;
    const float3 ray_direction = normalize(st.x * camera.u + st.y * camera.v - camera.w);

    // Pack payload pointer to 2 uints;
    float3 color;
    uint32_t i0, i1;
    packPointer(static_cast<void*>(&color), i0, i1);

    optixTrace(g_launch_params.handle,
               ray_origin,
               ray_direction,
               0.0f,
               1e16f,
               0.0f,
               OptixVisibilityMask{ 255 },
               OPTIX_RAY_FLAG_CULL_BACK_FACING_TRIANGLES | OPTIX_RAY_FLAG_DISABLE_ANYHIT,
               0,
               2,
               0,
               i0,
               i1);
    float4& accum_color  = g_launch_params.frame.accum_buffer[pixel_index];
    uchar4& output_color = g_launch_params.frame.color_buffer[pixel_index];
    
    if (accum_count > 0) {
        const float factor = 1.0f / static_cast<float>(accum_count + 1);
        color = lerp(make_float3(accum_color), color, factor);
    }

    accum_color  = make_float4(color, 1.0f);
    output_color = make_color(color);
}

extern "C" __global__ void __closesthit__radiance()
{
    const HitgroupData* data = reinterpret_cast<HitgroupData*>(optixGetSbtDataPointer());
    const HitResult result   = getHitResult(*data);

    float4 base_color = data->material->base_color * result.color;
    if (data->material->base_color_texture) {
        const float4 base_color_tex = sampleTexture<float4>(data->material->base_color_texture, result);
        const float3 base_color_tex_linear = linearize(make_float3(base_color_tex));
        base_color *= make_float4(base_color_tex_linear, base_color_tex.w);
    }

    const float cosdn = 0.2f + 0.8f * fabs(dot(optixGetWorldRayDirection(), result.normal));

    float3& payload = getPayload<float3&>();
    payload = cosdn * make_float3(base_color);
}

extern "C" __global__ void __miss__radiance()
{
    float3& color = getPayload<float3&>();
    color = g_launch_params.background_color;
}

extern "C" __global__ void __closesthit__occlusion()
{
}

extern "C" __global__ void __miss__occlusion()
{
}