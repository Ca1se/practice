#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>
#include <optix_device.h>

#include "Manet.h"
#include "random.h"
#include "Helper.h"
#include "Record.h"

extern "C" __constant__ LaunchParams g_launch_params;

extern "C" __global__ void __raygen__pinhole()
{
    const uint3    idx         = optixGetLaunchIndex();
    const uint3    dim         = optixGetLaunchDimensions();
    const size_t   pixel_index = dim.x * idx.y + idx.x;
    const uint32_t accum_count = g_launch_params.frame.accum_count;
    const auto&    camera      = g_launch_params.camera;

    // Get ray direction
    uint32_t seed = tea<4>(pixel_index, accum_count);
    const float2 subpixel_jitter = accum_count == 0
                                   ? make_float2(0.5f, 0.5f)
                                   : make_float2(rnd(seed), rnd(seed));
    const float2 st = 2.0f
                    * make_float2((static_cast<float>(idx.x) + subpixel_jitter.x) / static_cast<float>(dim.x),
                                  (static_cast<float>(idx.y) + subpixel_jitter.y) / static_cast<float>(dim.y))
                    - 1.0f;
    const float3 ray_origin    = camera.position;
    const float3 ray_direction = normalize(st.x * camera.u + st.y * camera.v + camera.w);

    // Pack payload pointer to 2 uints;
    float3 color;
    uint32_t i0, i1;
    packPointer(static_cast<void*>(&color), i0, i1);

    optixTrace(g_launch_params.handle,
               ray_origin,
               ray_direction,
               0.0f,
               1e16f,
               0.0f,
               OptixVisibilityMask{ 255 },
               OPTIX_RAY_FLAG_CULL_BACK_FACING_TRIANGLES | OPTIX_RAY_FLAG_DISABLE_ANYHIT,
               0,
               2,
               0,
               i0,
               i1);
    float4& accum_color  = g_launch_params.frame.accum_buffer[pixel_index];
    uchar4& output_color = g_launch_params.frame.color_buffer[pixel_index];
    
    if (accum_count > 0) {
        const float factor = 1.0f / static_cast<float>(accum_count + 1);
        color = lerp(make_float3(accum_color), color, factor);
    }

    accum_color  = make_float4(color, 1.0f);
    output_color = make_color(color);
}

extern "C" __global__ void __closesthit__radiance()
{
    const HitgroupData* data = reinterpret_cast<HitgroupData*>(optixGetSbtDataPointer());
    const uint32_t primitive_index = optixGetPrimitiveIndex();
    const float2 barycentrics = optixGetTriangleBarycentrics();
    const float b1 = barycentrics.x;
    const float b2 = barycentrics.y;
    const float b0 = 1.0f - b1 - b2;

    const uint3 index = data->indices[primitive_index];

    const float3 ray_origin    = optixGetWorldRayOrigin();
    const float3 ray_direction = optixGetWorldRayDirection();

    // position
    float3 intersection  = ray_origin + optixGetRayTmax() * ray_direction;
    intersection = optixTransformPointFromObjectToWorldSpace(intersection);
    
    // normal
    float3 normal;
    if (data->normals) {
        const float3 n0 = data->normals[index.x];
        const float3 n1 = data->normals[index.y];
        const float3 n2 = data->normals[index.z];
        normal = b0 * n0 + b1 * n1 + b2 * n2;
    } else {
        const float3 p0 = data->positions[index.x];
        const float3 p1 = data->positions[index.y];
        const float3 p2 = data->positions[index.z];
        normal = plainNormal(p0, p1, p2);
    }
    normal = optixTransformNormalFromObjectToWorldSpace(normal);
    normal = normalize(normal);

    // texcoord
    float3 texture_color;
    if (data->texcoords) {
        const float2 t0 = data->texcoords[index.x];
        const float2 t1 = data->texcoords[index.y];
        const float2 t2 = data->texcoords[index.z];
        const float2 texcoord = b0 * t0 + b1 * t1 + b2 * t2;
        texture_color = make_float3(tex2D<float4>(data->material->base_color_texture, texcoord.x, texcoord.y));
    } else {
        texture_color = make_float3(1.0f);
    }

    const float cosdn = 0.1f + 0.8f * fabs(dot(ray_direction, normal));
    const float3 color = make_float3(data->material->base_color) * texture_color * cosdn;

    // float3& payload = getPayload<float3>();
    // payload = color * 0.01f;
}

extern "C" __global__ void __miss__radiance()
{
    float3& color = getPayload<float3>();
    color = g_launch_params.background_color;
}

extern "C" __global__ void __closesthit__occlusion()
{
}

extern "C" __global__ void __miss__occlusion()
{
}