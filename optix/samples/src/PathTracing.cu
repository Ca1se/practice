#include "hip/hip_runtime.h"
#include "PathTracing.h"

#include <hip/hip_vector_types.h>
#include <optix_device.h>

#include "Manet.h"
#include "random.h"
#include "Helper.h"
#include "Record.h"
#include "HitResult.h"

namespace
{

__constant__ struct
{
    __forceinline__ __device__ OptixPayloadTypeID constexpr payloadType(PayloadType payload_type)
    {
        switch (payload_type) {
        case PAYLOAD_TYPE_RADIANCE:
            return OPTIX_PAYLOAD_TYPE_ID_0;
        case PAYLOAD_TYPE_OCCLUSION:
            return OPTIX_PAYLOAD_TYPE_ID_1;
        }
    }

} g_mapper;

__forceinline__ __device__ void setPayloadType(PayloadType payload_type)
{
    optixSetPayloadTypes(g_mapper.payloadType(payload_type));
}

__forceinline__ __device__ RadiancePayload& getRadiancePayload()
{
    setPayloadType(PAYLOAD_TYPE_RADIANCE);
    return getPayload<RadiancePayload&>();
}

__forceinline__ __device__ void
traceRadiance(OptixTraversableHandle handle,
              float                  tmin,
              float                  tmax,
              RadiancePayload&       payload)
{
    uint32_t u0, u1;
    packPointer(u0, u1, &payload);

    optixTrace(g_mapper.payloadType(PAYLOAD_TYPE_RADIANCE),
               handle,
               payload.ray_origin,
               payload.ray_direction,
               tmin,
               tmax,
               0.0f,  // ray time
               1,     // visibility mask
               OPTIX_RAY_FLAG_DISABLE_ANYHIT | OPTIX_RAY_FLAG_CULL_BACK_FACING_TRIANGLES,
               RAY_TYPE_RADIANCE,
               RAY_TYPE_COUNT,
               RAY_TYPE_RADIANCE,
               u0, u1);
}

__forceinline__ __device__ bool
traceOcclusion(OptixTraversableHandle handle,
               float3                 ray_origin,
               float3                 ray_direction,
               float                  tmin,
               float                  tmax)
{
    uint32_t occluded;
    optixTrace(g_mapper.payloadType(PAYLOAD_TYPE_OCCLUSION),
               handle,
               ray_origin,
               ray_direction,
               tmin,
               tmax,
               0.0f, // ray time
               1,    // visibility mask
               OPTIX_RAY_FLAG_DISABLE_ANYHIT | OPTIX_RAY_FLAG_TERMINATE_ON_FIRST_HIT,
               RAY_TYPE_OCCLUSION,
               RAY_TYPE_COUNT,
               RAY_TYPE_OCCLUSION,
               occluded);
    return occluded;
}

} // namespace

extern "C" __constant__ LaunchParams g_launch_params;

extern "C" __global__ void __raygen__pinhole()
{
    const uint3    idx               = optixGetLaunchIndex();
    const uint3    dim               = optixGetLaunchDimensions();
    const size_t   pixel_index       = dim.x * idx.y + idx.x;
    const uint32_t accum_count       = g_launch_params.frame.accum_count;
    const uint32_t samples_per_pixel = g_launch_params.samples_per_pixel;
    const uint32_t max_tracing_num   = g_launch_params.max_tracing_num;
    const auto&    camera            = g_launch_params.camera;

    uint32_t seed = tea<4>(pixel_index, accum_count);
    float3 result = make_float3(0.0f);

    for (uint32_t i = 0; i < samples_per_pixel; i++) {
        const float2 subpixel_jitter = make_float2(rnd(seed), rnd(seed));
        const float2 st = 2.0f
                        * make_float2((static_cast<float>(idx.x) + subpixel_jitter.x) / static_cast<float>(dim.x),
                                      (static_cast<float>(idx.y) + subpixel_jitter.y) / static_cast<float>(dim.y))
                        - 1.0f;
        RadiancePayload payload = {};
        payload.ray_origin    = camera.position;
        payload.ray_direction = normalize(st.x * camera.u + st.y * camera.v - camera.w);
        payload.seed          = seed;

        do {
            traceRadiance(g_launch_params.handle, 0.01f, 1e16f, payload);
            result += payload.attenuation * payload.radiance;
            payload.depth++;
        } while (!payload.done || payload.depth >= max_tracing_num);
    }

    float4& accum_color  = g_launch_params.frame.accum_buffer[pixel_index];
    uchar4& pixel_color  = g_launch_params.frame.color_buffer[pixel_index];
    float3  result_color = result / static_cast<float>(samples_per_pixel);

    if (accum_count > 0) {
        const float factor = 1.0f / static_cast<float>(accum_count + 1);
        result_color = lerp(make_float3(accum_color), result_color, factor);
    }

    accum_color = make_float4(result_color, 1.0f);
    pixel_color = make_color(result_color);
}

extern "C" __global__ void __closesthit__radiance()
{
    const HitgroupData* data        = reinterpret_cast<HitgroupData*>(optixGetSbtDataPointer());
    const HitResult     result      = getHitResult(*data);
    const PbrMaterial&  material    = *(data->material);
    const ParallelogramLight& light = g_launch_params.light;
    RadiancePayload&    payload  = getRadiancePayload();

    float4 base_color = material.base_color * result.color;
    if (material.base_color_texture) {
        const float4 texture_color        = sampleTexture<float4>(material.base_color_texture, result);
        const float3 texture_color_linear = linearize(make_float3(texture_color));
        base_color *= make_float4(texture_color_linear, texture_color.w);
    }

    float metallic  = material.metallic;
    float roughness = material.roughness;
    if (material.metallic_roughness_texture) {
        const float4 metallic_roughness = sampleTexture<float4>(material.metallic_roughness_texture, result);
        metallic  *= metallic_roughness.z;
        roughness *= metallic_roughness.y;
    }

    const float3 albedo         = make_float3(base_color);
    const float3 f0             = make_float3(0.04f);
    const float3 diffuse_color  = albedo * (make_float3(1.0f) - f0) * (1.0f - metallic);
    const float3 specular_color = lerp(f0, albedo, metallic);
    const float  alpha          = roughness * roughness;

    float3 radiance = make_float3(0.0f);
    float3 attenuation = make_float3(1.0f);

    // emission
    if (material.emissive_texture) {
        const float3 emissive_factor = material.emissive_factor;
        const float4 emissive_texture_color = sampleTexture<float4>(material.emissive_texture, result);
        radiance += emissive_factor * make_float3(emissive_texture_color);
    }

    // normal
    float3 normal = result.normal;
    if (material.normal_texture) {
        const float4 normal_sampled = 2.0f * sampleTexture<float4>(material.normal_texture, result) - make_float4(1.0f);
        const float2 rotation       = material.normal_texture.texcoord_rotation;
        const float2 tb             = make_float2(normal_sampled.x, normal_sampled.y);
        const float2 tb_trans       = make_float2(dot(tb, make_float2(rotation.y, -rotation.x)),
                                                  dot(tb, make_float2(rotation.x, rotation.y)));
        normal = normalize(tb_trans.x * result.texcoord.t + tb_trans.y * result.texcoord.b + normal_sampled.z * result.normal);
    }

    if (dot(normal, payload.ray_direction) > 0.0f)
        normal = -normal;
    
    // BRDF
    const float  s = 2.0f * rnd(payload.seed) - 1.0f;
    const float  t = 2.0f * rnd(payload.seed) - 1.0f;
    const float3 sample_pos = light.center + s * light.half_u + t * light.half_v;
    const float3 sample_dir = normalize(sample_pos - result.intersection);
    const float3 half_vec   = normalize(sample_dir - payload.ray_direction);
    const float  N_dot_L    = dot(normal, sample_dir);
    const float  N_dot_V    = dot(normal, -payload.ray_direction);
    const float  N_dot_H    = dot(normal, half_vec);
    const float  V_dot_H    = dot(-payload.ray_direction, half_vec);

    const float3 F    = schlick(specular_color, V_dot_H);
    const float  G    = smiths(N_dot_V, N_dot_L, roughness);
    const float  D    = ggxNormal(N_dot_H, alpha);
    const float3 f_r  = 0.25f * G * D / (N_dot_L * N_dot_V) * F;

    // direct light
    if (dot(normal, sample_dir) > 0.0f && dot(normal, -payload.ray_direction) > 0.0f) {
        const float distance = length(sample_pos - result.intersection);
        const bool  occluded = traceOcclusion(g_launch_params.handle,
                                                payload.ray_origin,
                                                payload.ray_direction,
                                                0.01f,
                                                distance - 0.01f);
                                            
        if (!occluded) {
            const float area = 4.0f * length(light.half_u) * length(light.half_v);
            radiance += light.emission * f_r * N_dot_L * dot(light.normal, -sample_dir) * area / (distance * distance);
        }
    }
    attenuation *= f_r * N_dot_L * 2.0f * MANET_PI / g_launch_params.p_rr;

    // uniform hemisphere sample
    
}

extern "C" __global__ void __miss__radiance()
{
    RadiancePayload& payload = getRadiancePayload();

    payload.radiance = make_float3(0.0f); // if depth > 0 and ray missed, then no radiance
    if (payload.depth == 0) // directly hit the background
        payload.radiance = g_launch_params.background_color;

    payload.done = true;
}

extern "C" __global__ void __closesthit__occlusion()
{
}

extern "C" __global__ void __miss__occlusion()
{
}